#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>



#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}

#define MATRIX_SIZE 1024

int BLOCK_WIDTH,BLOCK_HEIGHT;
void init_mat(float mat[MATRIX_SIZE][MATRIX_SIZE]);
static __global__ void cuda_1(float C[MATRIX_SIZE][MATRIX_SIZE],float A[MATRIX_SIZE][MATRIX_SIZE] , float B[MATRIX_SIZE][MATRIX_SIZE]);

extern "C" void cuda_version_1(float C[MATRIX_SIZE][MATRIX_SIZE],float A[MATRIX_SIZE][MATRIX_SIZE] , float B[MATRIX_SIZE][MATRIX_SIZE])
{

	void *A_dev, *B_dev, *C_dev;

	hipEvent_t start,stop;
	

	CUDA_CALL(hipMalloc((void**)&A_dev, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE));
	CUDA_CALL(hipMalloc((void**)&B_dev, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE));
	CUDA_CALL(hipMalloc((void**)&C_dev, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE));

	// take data from host to device.
	CUDA_CALL(hipMemcpy(A_dev, A, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE,hipMemcpyHostToDevice));

	CUDA_CALL(hipMemcpy(A_dev, B, sizeof(float) * MATRIX_SIZE * MATRIX_SIZE,hipMemcpyHostToDevice));

	dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT);
	dim3 grid(MATRIX_SIZE/BLOCK_WIDTH, MATRIX_SIZE/BLOCK_HEIGHT);
	float dev_time;



	printf("[CUDA 1] Start Launching Kernel.\n");
	//CHECK_TIME_START_GPU();

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	cuda_1<<<grid, block>>>((float(*)[MATRIX_SIZE])C_dev, (float(*)[MATRIX_SIZE])A_dev, (float(*)[MATRIX_SIZE])B_dev);

	//CHECK_TIME_END_GPU(dev_time);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&dev_time, start, stop);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("[CUDA 1] Processing Elapsed Time : %.3f (sec).\n",dev_time/1000);




	//take result from device to host.
	CUDA_CALL(hipMemcpy( C, C_dev, sizeof(float) * MATRIX_SIZE* MATRIX_SIZE, hipMemcpyDeviceToHost));
	CUDA_CALL( hipDeviceSynchronize() );


}
static __global__ void cuda_1(float C[MATRIX_SIZE][MATRIX_SIZE],float A[MATRIX_SIZE][MATRIX_SIZE] , float B[MATRIX_SIZE][MATRIX_SIZE])
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	float result = 0.0f;

	for(int i = 0 ; i < MATRIX_SIZE; ++i)
	{
		result += A[row][i] * B[i][col];
	}
	C[row][col] = result;


}

void init_mat(float mat[MATRIX_SIZE][MATRIX_SIZE])
{
	for (int i = 0 ; i < MATRIX_SIZE; ++i)
		for(int j = 0 ; j < MATRIX_SIZE; ++j)
			mat[i][j] = ( (float) rand()*2.f/RAND_MAX ) - 1.f;

}

float A[MATRIX_SIZE][MATRIX_SIZE],B[MATRIX_SIZE][MATRIX_SIZE],C_cuda_1[MATRIX_SIZE][MATRIX_SIZE];

int main(int argc, char *argv[])
{

	printf("input BLOCK WIDTH , BLOCK HEIGHT : ");
	scanf("%d %d",&BLOCK_WIDTH,&BLOCK_HEIGHT);
	init_mat(A);
	init_mat(B);

	cuda_version_1(C_cuda_1, A, B);


}
