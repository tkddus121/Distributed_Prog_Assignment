#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define ARRAY_SIZE 10000
#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
void init_array(int arr[ARRAY_SIZE])
{
	for(int i  = 0 ; i < ARRAY_SIZE ; ++i)
		arr[i] = rand()/10000;

}
static __global__ void cuda_noPath(int *arr, int *output, int size)
{
	int tid = blockIdx.x* blockDim.x + threadIdx.x;

	for(int i = 1 ; i < blockDim.x ; i *= 2)
	{
		if (threadIdx.x % (i*2) == 0 && tid + i < size)
			arr[tid] = max(arr[tid], arr[tid + i]);
		__syncthreads();
	}

	if( threadIdx.x == 0)
		output[blockIdx.x] = arr[tid];
}

static __global__ void cuda_Path(int *arr, int *output, int size)
{
	int tid = blockIdx.x* blockDim.x + threadIdx.x;
	int base = blockIdx.x * blockDim.x;

	for(int i = 1 ; i < blockDim.x ; i *= 2)
	{
		int off = threadIdx.x * i*2;
		int idx = base + off;

		if ( off < blockDim.x  &&  idx + i < size)
			arr[tid] = max(arr[idx], arr[idx + i]);
		__syncthreads();
	}

	if( threadIdx.x == 0)
		output[blockIdx.x] = arr[tid];
}


extern "C" void cuda_2(int *res, int arr[ARRAY_SIZE] )
{
	void *output_dev, *arr_dev;

	int thread_num = 256;
	int size = ARRAY_SIZE;
	int block_num = (size + thread_num - 1) / thread_num;


	hipEvent_t start,stop;
	
    CUDA_CALL(hipMalloc((void**)&arr_dev, sizeof(int) * ARRAY_SIZE));
    CUDA_CALL(hipMalloc((void**)&output_dev, sizeof(int) * block_num));

    // transfer data from host to device.
    CUDA_CALL(hipMemcpy(arr_dev, arr, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice));

    float dev_time = 0.f;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
    do {

        //CHECK_TIME_START_GPU();
        cuda_Path<<<block_num, thread_num>>>((int*)arr_dev, (int*)output_dev, size);
        //CHECK_TIME_END_GPU(device_time);

        {
            void *tmp = arr_dev;
            arr_dev = output_dev;
            output_dev = tmp;
        }
        size = block_num;
        block_num = (size + thread_num - 1) / thread_num;

    } while (size > 1);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&dev_time, start, stop);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);

    printf("[CUDA Path] Elapsed Time : %.5f (sec).\n", dev_time/1000);

    // transfer result from device to host.
    CUDA_CALL(hipMemcpy(res, arr_dev, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CALL( hipDeviceSynchronize() );

    CUDA_CALL(hipFree(arr_dev));
    CUDA_CALL(hipFree(output_dev));


}

extern "C" void cuda_1(int *res, int arr[ARRAY_SIZE] )
{
	void *output_dev, *arr_dev;

	int thread_num = 256;
	int size = ARRAY_SIZE;
	int block_num = (size + thread_num - 1) / thread_num;


	hipEvent_t start,stop;
	
    CUDA_CALL(hipMalloc((void**)&arr_dev, sizeof(int) * ARRAY_SIZE));
    CUDA_CALL(hipMalloc((void**)&output_dev, sizeof(int) * block_num));

    // transfer data from host to device.
    CUDA_CALL(hipMemcpy(arr_dev, arr, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice));

    float dev_time = 0.f;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
    do {

        //CHECK_TIME_START_GPU();
        cuda_noPath<<<block_num, thread_num>>>((int*)arr_dev, (int*)output_dev, size);
        //CHECK_TIME_END_GPU(device_time);

        {
            void *tmp = arr_dev;
            arr_dev = output_dev;
            output_dev = tmp;
        }
        size = block_num;
        block_num = (size + thread_num - 1) / thread_num;

    } while (size > 1);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&dev_time, start, stop);
	
	hipEventDestroy(start);
	hipEventDestroy(stop);

    printf("[CUDA noPath] Elapsed Time : %.5f (sec).\n", dev_time/1000);

    // transfer result from device to host.
    CUDA_CALL(hipMemcpy(res, arr_dev, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CALL( hipDeviceSynchronize() );

    CUDA_CALL(hipFree(arr_dev));
    CUDA_CALL(hipFree(output_dev));


}
int arr[ARRAY_SIZE],
	res_cuda1,
	res_cuda2;

int main(int argc, char *argv[])
{

	init_array(arr);
	cuda_1(&res_cuda1,arr);
	cuda_2(&res_cuda2,arr);
	return 0;

}
